#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <string>
#include "TextHandler.h"
#include "MultiplierHandler.h"
#include "PatternHandler.h"

#define THREAD_COUNT 10
#define DIM_COUNT 10
#define MAX_PATTERN_LENGTH 20
#define MULTIPLIER 1117
#define MODULUS 2147483647

using namespace std;

void init(){}

TextHandler* handleText() {
  TextHandler* handler = new TextHandler(MULTIPLIER, MODULUS);
  handler->read();
  handler->preprocess();
  handler->share();
  return handler;
}

int32_t* handleMultipliers() {
  MultiplierHandler* handler = new MultiplierHandler(MULTIPLIER, MODULUS, MAX_PATTERN_LENGTH);
  handler->calculate();
  handler->share();
  return handler->dMultipliers;
}

PatternHandler* handlePatterns(size_t maxPatterns) {
  PatternHandler* handler = new PatternHandler();
  handler->read(maxPatterns);
  handler->share();
  return handler;
}

__global__ void searchForPatterns(
		int32_t *shas, int32_t shasLength,
		int32_t* multipliers, char* mergedPatterns,
		int32_t *patternStarts, int32_t* patternEnds,
		int32_t patternsCount) {

    int32_t idx = THREAD_COUNT * blockIdx.x + threadIdx.x;
    int64_t resultAtPosition;
    int64_t sha;
    // printf("[%d, %d] will work starting from: %d", blockIdx.x, threadIdx.x, idx);

    while (idx < patternsCount) {
      sha = 0;
      const int32_t length = patternEnds[idx] - patternStarts[idx];
      for (int32_t i = patternStarts[idx]; i<patternEnds[idx]; i++) {
        sha = (sha * MULTIPLIER + mergedPatterns[i]) % MODULUS;
      }
      if (shas[length - 1] == (int32_t)sha) {
        printf("Possible match at 0 for %d\n", idx);
      }
      for (int32_t i = length; i<shasLength; i++) {
        resultAtPosition = -shas[i - length];
        resultAtPosition *= multipliers[length];
        resultAtPosition += shas[i];
        resultAtPosition %= MODULUS;
        resultAtPosition = resultAtPosition < 0 ? resultAtPosition + MODULUS : resultAtPosition;
       if (resultAtPosition == sha) {
          printf("Possible match at %d for %d\n", i - length + 1, idx);
       }
      }
      idx += DIM_COUNT * THREAD_COUNT;;
    }
}

void calculate(int32_t *preprocessed, int32_t preprocessedLength, int32_t* multipliers, char* mergedPatterns,
  int32_t *patternStarts, int32_t* patternEnds, int32_t patternsCount) {
  searchForPatterns<<<DIM_COUNT, THREAD_COUNT>>>(
          preprocessed, preprocessedLength,
          multipliers,
          mergedPatterns, patternStarts, patternEnds, patternsCount
        );
}

void join() {
  hipDeviceSynchronize();
  hipError_t code = hipGetLastError();
  if (code != hipSuccess)
  {
          fprintf(stderr,"Cuda call failed: [%d] %s\n", code, hipGetErrorString(code));
          exit(code);
  }
}

int main(int argc, char **argv) {
  size_t maxPatterns = 100000000;
  if (argc > 1) {
    maxPatterns = atoi(argv[1]);
  }
  init();

  TextHandler* preprocessed = handleText();
  int32_t* multipliers = handleMultipliers();
  PatternHandler* patterns = handlePatterns(maxPatterns);
  int patternCount = min(patterns -> count, maxPatterns);
  printf("Will work on %d patterns\n", patternCount);
  calculate(preprocessed -> dShas, preprocessed -> length, multipliers,
    patterns -> dMerged, patterns -> dStarts, patterns -> dEnds, patternCount
  );

  join();
}
